#include "hip/hip_runtime.h"
#include "cuda_gl_interop.h"
#include "cuda_check_error.h"

#include "SFML/Graphics.hpp"
#include "SFML/Graphics/Image.hpp"

#define THREADS_PER_BLOCK 256

#define WIDTH 1024
#define HEIGHT 768

__global__ void update_surface(hipSurfaceObject_t surface)
{
  int x = threadIdx.x + blockIdx.x * blockDim.x;
  int y = threadIdx.y + blockIdx.y * blockDim.y;

  if(x >= WIDTH)
    return;

  uchar4 pixel = { x & 0xff, x & 0xff, x & 0xff, 0xff };

  surf2Dwrite(pixel, surface, x * sizeof(uchar4), y);
}

int main(int argc, char **argv)
{
  sf::RenderWindow window(sf::VideoMode(WIDTH, HEIGHT), "cuda_gl_interop");

  sf::Sprite sprite;
  sf::Texture txture;
  txture.create(WIDTH, HEIGHT);
  
  hipArray *bitmap_d;

  GLuint gl_tex_handle = txture.getNativeHandle();

  hipGraphicsResource *cuda_tex_handle;

  hipGraphicsGLRegisterImage(&cuda_tex_handle, gl_tex_handle, GL_TEXTURE_2D,
                              hipGraphicsRegisterFlagsNone);
  cudaCheckError();

  hipGraphicsMapResources(1, &cuda_tex_handle, 0);
  cudaCheckError();

  hipGraphicsSubResourceGetMappedArray(&bitmap_d, cuda_tex_handle, 0, 0);
  cudaCheckError();

  struct hipResourceDesc resDesc;
  memset(&resDesc, 0, sizeof(resDesc));
  resDesc.resType = hipResourceTypeArray;

  resDesc.res.array.array = bitmap_d;
  hipSurfaceObject_t bitmap_surface = 0;
  hipCreateSurfaceObject(&bitmap_surface, &resDesc);
  cudaCheckError();

  sprite.setTexture(txture);

  dim3 blocks(ceil((float)WIDTH / THREADS_PER_BLOCK), HEIGHT);

  while(!sf::Keyboard::isKeyPressed(sf::Keyboard::Escape))
  {
    update_surface<<<blocks, THREADS_PER_BLOCK>>>(bitmap_surface);
    cudaCheckError();

    hipDeviceSynchronize();
    cudaCheckError();

    window.draw(sprite);
    window.display();
  }

  return 0;
}
